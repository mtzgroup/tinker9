#include "hip/hip_runtime.h"
#include "epolar.h"
#include "field.h"
#include "induce.h"
#include "launch.h"
#include "md.h"
#include "mod.uprior.h"
#include "qmmm_global.h"
#include "tool/cudalib.h"
#include "tool/io_print.h"
#include <tinker/detail/inform.hh>
#include <tinker/detail/polpcg.hh>
#include <tinker/detail/polpot.hh>
#include <tinker/detail/units.hh>


namespace tinker {
#define ITHREAD threadIdx.x + blockIdx.x* blockDim.x
#define STRIDE  blockDim.x* gridDim.x


__global__
void pcg_udir(int n, const real* restrict polarity, real (*restrict udir)[3],
              real (*restrict udirp)[3], const real (*restrict field)[3],
              const real (*restrict fieldp)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real poli = polarity[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         udir[i][j] = poli * field[i][j];
         udirp[i][j] = poli * fieldp[i][j];
      }
   }
}


__global__
void pcg_rsd(int n, const real* restrict polarity_inv,         //
             real (*restrict rsd)[3], real (*restrict rsp)[3], //
             const real (*restrict udir)[3], const real (*restrict udip)[3],
             const real (*restrict uind)[3], const real (*restrict uinp)[3],
             const real (*restrict field)[3], const real (*restrict fielp)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real poli_inv = polarity_inv[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         rsd[i][j] = (udir[i][j] - uind[i][j]) * poli_inv + field[i][j];
         rsp[i][j] = (udip[i][j] - uinp[i][j]) * poli_inv + fielp[i][j];
      }
   }
}


__global__
void pcg_rsd0(int n, const real* restrict polarity, real (*restrict rsd)[3],
              real (*restrict rsdp)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      if (polarity[i] == 0) {
         rsd[i][0] = 0;
         rsd[i][1] = 0;
         rsd[i][2] = 0;
         rsdp[i][0] = 0;
         rsdp[i][1] = 0;
         rsdp[i][2] = 0;
      }
   }
}


__global__
void pcg_p1(int n, const real* restrict polarity_inv, real (*restrict vec)[3],
            real (*restrict vecp)[3], const real (*restrict conj)[3],
            const real (*restrict conjp)[3], const real (*restrict field)[3],
            const real (*restrict fieldp)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real poli_inv = polarity_inv[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         vec[i][j] = poli_inv * conj[i][j] - field[i][j];
         vecp[i][j] = poli_inv * conjp[i][j] - fieldp[i][j];
      }
   }
}


__global__
void pcg_p2(int n, const real* restrict polarity,              //
            const real* restrict ka, const real* restrict kap, //
            const real* restrict ksum, const real* restrict ksump,
            real (*restrict uind)[3], real (*restrict uinp)[3],
            const real (*restrict conj)[3], const real (*restrict conjp)[3],
            real (*restrict rsd)[3], real (*restrict rsdp)[3],
            const real (*restrict vec)[3], const real (*restrict vecp)[3])
{
   real kaval = *ka, kapval = *kap;
   real a = *ksum / kaval, ap = *ksump / kapval;
   if (kaval == 0)
      a = 0;
   if (kapval == 0)
      ap = 0;
   for (int i = ITHREAD; i < n; i += STRIDE) {
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         uind[i][j] += a * conj[i][j];
         uinp[i][j] += ap * conjp[i][j];
         rsd[i][j] -= a * vec[i][j];
         rsdp[i][j] -= ap * vecp[i][j];
      }
      if (polarity[i] == 0) {
         rsd[i][0] = 0;
         rsd[i][1] = 0;
         rsd[i][2] = 0;
         rsdp[i][0] = 0;
         rsdp[i][1] = 0;
         rsdp[i][2] = 0;
      }
   }
}


__global__
void pcg_p3(int n, const real* restrict ksum, const real* restrict ksump,
            const real* restrict ksum1, const real* restrict ksump1,
            real (*restrict conj)[3], real (*restrict conjp)[3],
            real (*restrict zrsd)[3], real (*restrict zrsdp)[3])
{
   real kaval = *ksum, kapval = *ksump;
   real b = *ksum1 / kaval, bp = *ksump1 / kapval;
   if (kaval == 0)
      b = 0;
   if (kapval == 0)
      bp = 0;
   for (int i = ITHREAD; i < n; i += STRIDE) {
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         conj[i][j] = zrsd[i][j] + b * conj[i][j];
         conjp[i][j] = zrsdp[i][j] + bp * conjp[i][j];
      }
   }
}


__global__
void pcg_peek(int n, float pcgpeek, const real* restrict polarity,
              real (*restrict uind)[3], real (*restrict uinp)[3],
              const real (*restrict rsd)[3], const real (*restrict rsdp)[3])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real term = pcgpeek * polarity[i];
      #pragma unroll
      for (int j = 0; j < 3; ++j) {
         uind[i][j] += term * rsd[i][j];
         uinp[i][j] += term * rsdp[i][j];
      }
   }
}


void induce_mutual_pcg1_cu(real (*uind)[3], real (*uinp)[3])
{
   auto* field = work01_;
   auto* fieldp = work02_;
   auto* rsd = work03_;
   auto* rsdp = work04_;
   auto* zrsd = work05_;
   auto* zrsdp = work06_;
   auto* conj = work07_;
   auto* conjp = work08_;
   auto* vec = work09_;
   auto* vecp = work10_;


   const bool sparse_prec = polpcg::pcgprec;
   bool dirguess = polpcg::pcgguess;
   bool predict = polpred != UPred::NONE;
   if (predict and nualt < maxualt) {
      predict = false;
      dirguess = true;
   }


   // get the electrostatic field due to permanent multipoles
   if (QMMMGlobal::if_replace_electric_field_for_compute_induced_dipole
       && QMMMGlobal::n_qm > 0)
   {
      darray::copy(g::q0, n, field, QMMMGlobal::d_qmmm_electric_field_d);
      darray::copy(g::q0, n, fieldp, QMMMGlobal::d_qmmm_electric_field_p);
   }
   else
      dfield(field, fieldp);
   // direct induced dipoles
   launch_k1s(g::s0, n, pcg_udir, n, polarity, udir, udirp, field, fieldp);


   // initial induced dipole
   if (predict) {
      ulspred_sum(uind, uinp);
   } else if (dirguess) {
      darray::copy(g::q0, n, uind, udir);
      darray::copy(g::q0, n, uinp, udirp);
   } else {
      darray::zero(g::q0, n, uind, uinp);
   }


   // initial residual r(0)
   //
   // if use pcgguess, r(0) = E - (inv_alpha + Tu) alpha E
   //                       = E - E -Tu udir
   //                       = -Tu udir
   //
   // in general, r(0) = E - (inv_alpha + Tu) u(0)
   //                  = -Tu u(0) + E - inv_alpha u(0)
   //                  = -Tu u(0) + inv_alpha (udir - u(0))
   //
   // if do not use pcgguess, r(0) = E - T Zero = E
   if (predict) {
      ufield(uind, uinp, field, fieldp);
      launch_k1s(g::s0, n, pcg_rsd, n, polarity_inv, rsd, rsdp, udir, udirp,
                 uind, uinp, field, fieldp);
   } else if (dirguess) {
      ufield(udir, udirp, rsd, rsdp);
   } else {
      darray::copy(g::q0, n, rsd, field);
      darray::copy(g::q0, n, rsdp, fieldp);
   }
   launch_k1s(g::s0, n, pcg_rsd0, n, polarity, rsd, rsdp);


   // initial M r(0) and p(0)
   if (sparse_prec) {
      sparse_precond_build();
      sparse_precond_apply(rsd, rsdp, zrsd, zrsdp);
   } else {
      diag_precond(rsd, rsdp, zrsd, zrsdp);
   }
   darray::copy(g::q0, n, conj, zrsd);
   darray::copy(g::q0, n, conjp, zrsdp);


   // initial r(0) M r(0)
   real* sum = &((real*)dptr_buf)[0];
   real* sump = &((real*)dptr_buf)[1];
   darray::dot(g::q0, n, sum, rsd, zrsd);
   darray::dot(g::q0, n, sump, rsdp, zrsdp);


   // conjugate gradient iteration of the mutual induced dipoles
   const bool debug = inform::debug;
   const int politer = polpot::politer;
   const real poleps = polpot::poleps;
   const real debye = units::debye;
   const real pcgpeek = polpcg::pcgpeek;
   const int maxiter = 100; // see also subroutine induce0a in induce.f


   bool done = false;
   int iter = 0;
   real eps = 100;
   real epsold;


   while (!done) {
      ++iter;


      // T p and p
      // vec = (inv_alpha + Tu) conj, field = -Tu conj
      // vec = inv_alpha * conj - field
      ufield(conj, conjp, field, fieldp);
      launch_k1s(g::s0, n, pcg_p1, n, polarity_inv, vec, vecp, conj, conjp,
                 field, fieldp);


      // a <- p T p
      real* a = &((real*)dptr_buf)[2];
      real* ap = &((real*)dptr_buf)[3];
      // a <- r M r / p T p; a = sum / a; ap = sump / ap
      darray::dot(g::q0, n, a, conj, vec);
      darray::dot(g::q0, n, ap, conjp, vecp);


      // u <- u + a p
      // r <- r - a T p
      launch_k1s(g::s0, n, pcg_p2, n, polarity, a, ap, sum, sump, uind, uinp,
                 conj, conjp, rsd, rsdp, vec, vecp);


      // calculate/update M r
      if (sparse_prec)
         sparse_precond_apply(rsd, rsdp, zrsd, zrsdp);
      else
         diag_precond(rsd, rsdp, zrsd, zrsdp);


      // b = sum1 / sum; bp = sump1 / sump
      real* sum1 = &((real*)dptr_buf)[4];
      real* sump1 = &((real*)dptr_buf)[5];
      darray::dot(g::q0, n, sum1, rsd, zrsd);
      darray::dot(g::q0, n, sump1, rsdp, zrsdp);


      // calculate/update p
      launch_k1s(g::s0, n, pcg_p3, n, sum, sump, sum1, sump1, conj, conjp, zrsd,
                 zrsdp);


      // copy sum1/p to sum/p
      darray::copy(g::q0, 2, sum, sum1);


      real* epsd = &((real*)dptr_buf)[6];
      real* epsp = &((real*)dptr_buf)[7];
      darray::dot(g::q0, n, epsd, rsd, rsd);
      darray::dot(g::q0, n, epsp, rsdp, rsdp);
      check_rt(hipMemcpyAsync((real*)pinned_buf, epsd, 2 * sizeof(real),
                               hipMemcpyDeviceToHost, g::s0));
      check_rt(hipStreamSynchronize(g::s0));
      epsold = eps;
      eps = REAL_MAX(((real*)pinned_buf)[0], ((real*)pinned_buf)[1]);
      eps = debye * REAL_SQRT(eps / n);


      if (debug) {
         if (iter == 1) {
            print(stdout,
                  "\n Determination of SCF Induced Dipole Moments\n\n"
                  "    Iter    RMS Residual (Debye)\n\n");
         }
         print(stdout, " %8d       %-16.10f\n", iter, eps);
      }


      if (eps < poleps)
         done = true;
      if (eps > epsold)
         done = true;
      if (iter >= politer)
         done = true;


      // apply a "peek" iteration to the mutual induced dipoles
      if (done) {
         launch_k1s(g::s0, n, pcg_peek, n, pcgpeek, polarity, uind, uinp, rsd,
                    rsdp);
      }
   }


   // print the results from the conjugate gradient iteration
   if (debug) {
      print(stdout,
            " Induced Dipoles :    Iterations %4d      RMS "
            "Residual %14.10f\n",
            iter, eps);
   }


   // terminate the calculation if dipoles failed to converge
   if (iter >= maxiter || eps > epsold) {
      prterr();
      TINKER_THROW("INDUCE  --  Warning, Induced Dipoles are not Converged");
   }
}
}
